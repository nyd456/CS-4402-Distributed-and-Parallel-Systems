﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <cassert>
#include <ctime>
#include <vector>

using namespace std;

/* ******************************
*  Exception handler functions
*
* Reference: sample_example
****************************** */
struct cuda_exception
{
	explicit cuda_exception(const char* err) : error_info(err) {}
	explicit cuda_exception(const string& err) : error_info(err) {}
	string what() const throw() { return error_info; }

private:
	string error_info;
};

void checkCudaError(const char* msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		string error_info(msg);
		error_info += " : ";
		error_info += hipGetErrorString(err);
		throw cuda_exception(error_info);
	}
}

/**
* CUDA kernel for univariate polynomial multiplication
*
* @a, the first input coefficients of polynomial
* @b, the second input coefficients of polynomial
* @c, the output coefficients of polynomial
* @n, degree of polynomial
*
* reference: Dependence_Analysis_and_Parallelization.pdf (slide 41)
*/
__global__ void polynomials_mul(int* a, int* b, int* c, int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < 2 * n + 1)
	{
		c[idx] = 0;
		for (int t = max(0, idx - n); t <= min(idx, n); t++)
		{
			c[idx] = c[idx] + a[t] * b[idx - t];
		}
	}
}

/*
* Check result on the CPU (C function)
*/
void verify_result(int* a, int* b, int* c, int n)
{
	clock_t t1 = clock();
	for (int i = 0; i < 2 * n + 1; i++)
	{
		int k = i;
		int result = 0;
		for (int j = 0; j <= k; j++)
		{
			if (j <= n && k - j <= n)
			{
				result = result + a[j] * b[k - j];
			}
		}
		assert(result == c[i]);
	}
	clock_t t2 = clock();
	cout << "   C Function Verification Passed! - takes " << (t2 - t1) / double(CLOCKS_PER_SEC) * 1000 << " ms" << endl;
}

/*
* Initialize c with random coefficients from {-1, 0, 1}
*
* @c, :an array of integers representing the coefficients of a polynomial
*  that is randomly generated with coefficients chosen from the set {-1, 0, 1}
*
*   n: the size of polynomial terms
*/
void random_polynomial(int* c, int n)
{
	srand(time(NULL));
	for (int i = 0; i < n; i++) {
		c[i] = rand() % 3 - 1;  // Generate random coefficient from {-1, 0, 1}
	}
}

/*
*  print out passed polynomial of size n
*
* c: coefficients of polynomial
* n: size of polynomial terms

*/
void print_polynomial(int* c, int n)
{
	for (int i = 0; i < n; i++)
	{
		cout << "   " << c[i] << " ";
	}
	cout << endl;
}
/*
*  Testing functon
*	Retun the testing running time for tracking the performance
*
*	@e, the exponent
*	@B, the threads per block
*/
int run_test(int e, int B) {
	double time;
	cout << "   --------------------------------------------------" << endl;
	cout << "   Case: B=" << B << " and n=" << "2^" << e;

	clock_t t1 = clock();
	size_t n = 1ULL << e; // calculate n based on the exponent
	size_t bytes = (n + 1) * sizeof(int); // size of a and b
	size_t bytes_c = (2 * n + 1) * sizeof(int); //  size of c

	int* a_h, * b_h, * c_h; // host arrays
	int* a_d, * b_d, * c_d; // device arrays
	try
	{
		// allocate memory
		a_h = (int*)malloc(bytes);
		b_h = (int*)malloc(bytes);
		c_h = (int*)malloc(bytes_c);

		// Initialize host random coefficients from {-1, 0, 1}
		random_polynomial(a_h, n + 1);
		random_polynomial(b_h, n + 1);

		// initialize polynomial c_h with size 2n+1
		int cSize = 2 * n + 1;
		for (int i = 0; i < cSize; i++)
		{
			c_h[i] = 0;
		}

		// Allocate device memory
		hipMalloc((void**)&a_d, bytes);
		hipMalloc((void**)&b_d, bytes);
		hipMalloc((void**)&c_d, bytes_c);

		// Copy data to the device
		hipMemcpy(a_d, a_h, bytes, hipMemcpyHostToDevice);
		hipMemcpy(b_d, b_h, bytes, hipMemcpyHostToDevice);

		// compute the execution configure
		// number of blocks (round up to the nearest whole number of blocks)
		int nBlocks = (2 * n + 1) / B + ((2 * n + 1) % B == 0 ? 0 : 1);

		// block size
		int bSize = B;

		polynomials_mul <<<nBlocks, bSize >>> (a_d, b_d, c_d, n);
		hipDeviceSynchronize();

		// read c from the device 
		hipMemcpy(c_h, c_d, bytes_c, hipMemcpyDeviceToHost);

		clock_t t2 = clock();
		time = (t2 - t1) / double(CLOCKS_PER_SEC) * 1000;
		cout << " takes " << time << " ms\n" << endl;

		// print polynomial
		if (n < 10)
		{
			cout << "   Input array a:" << endl;
			print_polynomial(a_h, n + 1);
			cout << "   Input array b:" << endl;
			print_polynomial(b_h, n + 1);
			cout << endl;
			cout << "   Output array c:" << endl;
			print_polynomial(c_h, 2 * n + 1);
		}

		//Check result with C function
		verify_result(a_h, b_h, c_h, n);
	}
	catch (cuda_exception& err)
	{
		cout << err.what() << endl;
		hipFree(a_d);
		hipFree(b_d);
		hipFree(c_d);

		free(a_h);
		free(b_h);
		free(c_h);
		return EXIT_FAILURE;
	}
	catch (...)
	{
		cout << "unknown exeception" << endl;
		hipFree(a_d);
		hipFree(b_d);
		hipFree(c_d);

		free(a_h);
		free(b_h);
		free(c_h);
		return EXIT_FAILURE;
	}


	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	free(a_h);
	free(b_h);
	free(c_h);

	cout << "   --------------------------------------------------" << endl;
	return time;
}


/**
*	Implement a CUDA program using ⌈(2n + 1)/B⌉
*	 thread-blocks with B = 32 threads.
*/
void run_question_1() {
	int exponent;
	cout << "\n   ***************   Question 1   ***************" << endl;
		cout << "   The polynomial degree n = 2^exponent\n" << endl;
		cout << "   The input/output coefficients will be \n   printed out only if n < 10 (exponent < 4)\n" << endl;
		cout << "   Thread per block B = 32" << endl;
		cout << "   Defaut exponent = 16 if input value < 0" << endl;
		cout << "   ********************************************* " << endl;

	while (true) {
		cout << "   Please input an exponent or type a non-number to exit: ";
		if (!(cin >> exponent)) {
			// Input is not a number, clear input buffer and exit loop
			cin.clear();
			cin.ignore(numeric_limits<streamsize>::max(), '\n');
			break;
		}

		cout << endl;

		// Default exponent is 16 if input is less than or equal to 1
		int e = (exponent < 0) ? 16 : exponent;
		int B = 32; // thread-blocks

		run_test(e, B);
	}
}

void run_question_2()
{
	cout << "\n   ***************   Question 2   ***************" << endl;
	int exponents[] = { 14, 16 };
	int Bs[] = { 32, 64, 128, 256, 512 };
	int e_len = sizeof(exponents) / sizeof(exponents[0]);
	int b_len = sizeof(Bs) / sizeof(Bs[0]);

	vector<pair<int, int>> bestPerformances(e_len, { numeric_limits<int>::max(), 0 });

	for (int i = 0; i < e_len; i++)
	{
		int e = exponents[i];

		for (int j = 0; j < b_len; j++)
		{
			int B = Bs[j];

			int time = run_test(e, B);

			// Update best performance B for this exponent if needed
			if (time < bestPerformances[i].first)
			{
				bestPerformances[i].first = time;
				bestPerformances[i].second = B;
			}
		}
	}
	cout << "\n   Best performance B for n = 2^" << exponents[0] << ": " << bestPerformances[0].second << endl;
	cout << "   Best performance B for n = 2^" << exponents[1] << ": " << bestPerformances[1].second << endl;
}

int main(int argc, char** argv)
{
	int input;
	if (argc >= 2)
	{
		input = atoi(argv[1]);
	}
	else
	{
		cout << "\n  Enter 1 for Question 1 or 2 for Question 2: ";
		scanf("%d", &input);
		cout << endl;

		// Consume the newline character from the input buffer
		while (getchar() != '\n');

		while (input != 1 && input != 2)
		{
			cout << "\n  Invalid input. Enter 1 for Question 1 or 2 for Question 2: ";
			scanf("%d", &input);
			cout << endl;

			// Consume the newline character from the input buffer
			while (getchar() != '\n');
		}
	}

	switch (input)
	{
	case 1:
		run_question_1();
		break;
	case 2:
		run_question_2();
		break;
	default:
		printf("Invalid number. Running Question 1 by default.\n");
		run_question_1();
		break;
	}

	return 0;
}
